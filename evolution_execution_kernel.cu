#include "hip/hip_runtime.h"
/*
    Pcuda: Simulating P systems with active membranes on the GPU 
    This simulator is published on:
    J.M. Cecilia, J.M. García, G.D. Guerrero, M.A. Martínez-del-Amor, I. Pérez-Hurtado,
    M.J. Pérez-Jiménez. Simulation of P systems with active membranes on CUDA,
    Briefings in Bioinformatics, 11, 3 (2010), 313-322

    Pcuda is a subproject of PMCGPU (Parallel simulators for Membrane 
                                       Computing on the GPU)   
 
    Copyright (c) 2009 Miguel Á. Martínez-del-Amor (RGNC, University of Seville)
 		       Ginés D. Guerrero (GACOP, University of Murcia)
		       Chema Cecilia (GACOP, University of Murcia)
		       Ignacio Pérez-Hurtado (RGNC, University of Seville)
    
    This file is part of Pcuda.
  
    Pcuda is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    Pcuda is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with Pcuda.  If not, see <http://www.gnu.org/licenses/>. */


#if !defined(__EVOLUTION_EXECUTION_KERNEL_CU__)
#define __EVOLUTION_EXECUTION_KERNEL_CU__

#include "pcuda_types.h"
#include "p_system.h"

__global__ void evolution_execution_kernel(const Rodruleset rules, const Rodmultiset objEvo,
                                           const Membraneset membranes, const uint numMemb, 
                                           const uint numLabs, const ushort numObjects,
                                           ushort * multisets, ushort * rev) {
    __shared__ short label;
    __shared__ short charge;

    const short objsPerThread = numObjects/blockDim.x;
    const uint lid = threadIdx.x * objsPerThread;
    const uint bid = blockIdx.y * gridDim.x + blockIdx.x;
    const uint tid = bid * numObjects + lid;
    ushort multiplicity;
    uint begin;
    ushort len;
    uint max;

    if (bid >= numMemb)
	return;
    
    label = membranes[bid].label;
    charge = membranes[bid].charge;

    if (label == EMPTY_MEMBRANE)
        return;

    #pragma unroll
    for (uint obj=0; obj < objsPerThread; obj++) {
        multiplicity = rev[tid + obj];

        if (multiplicity > 0) {
            begin = rules[(lid + obj) * numLabs * NUMBER_OF_CHARGES + label * NUMBER_OF_CHARGES + charge].rev & 0x000FFFFF;
            len = (rules[(lid + obj) * numLabs * NUMBER_OF_CHARGES + label * NUMBER_OF_CHARGES + charge].rev&0xFFF00000)>>20;
            max = begin + len;

            #pragma unroll
            for (uint i=begin; i < max; i++)
                multisets[bid * numObjects + objEvo[i].obj] += objEvo[i].mult * multiplicity;
                //atomicAdd((uint *)&multisets[bid * numObjects + objEvo[i].obj], (uint)(objEvo[i].mult * multiplicity));
        }
    }
}

#endif	/* __EVOLUTION_EXECUTION_KERNEL_CU__ */
